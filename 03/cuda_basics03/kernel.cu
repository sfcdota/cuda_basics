#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <stdio.h>
#include "sm_35_atomic_functions.h"
#include <iostream>
#include <math.h>
#include <time.h>
#include "hiprand/hiprand_kernel.h"
#define BLOCKS_LIMIT 127
#define EXPERIMENTS_LIMIT 100000
#define PI_KEK 3.141592653589793238462643383279502884

__global__ void setStates(hiprandState* states, long long dots, time_t seed)
{
	int id = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
	hiprand_init(seed, id, 0, &states[id]);
	hiprand_init(seed + 1, id, 0, &states[id + 1]);

}

__device__ float rand_float(hiprandState *state)
{
	return hiprand_uniform(state);
}

__global__ void ineffectiveMonteCarloCall( unsigned long long *hits, hiprandState *states, int experiments)
{
	int id = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
	for (int i = 0; i < experiments; i++)
		if (pow(rand_float(&states[id]), 2) + pow(rand_float(&states[id + 1]), 2) <= 1.0)
			atomicAdd(hits, 1ull);
}


__global__ void effectiveMonteCarloCall(unsigned long long* hits, hiprandState* states, int experiments)
{
	int id = 2 * (threadIdx.x + blockIdx.x * blockDim.x);
	int res = 0;
	for (int i = 0; i < experiments; i++)
		if (pow(rand_float(&states[id]), 2) + pow(rand_float(&states[id + 1]), 2) <= 1.0)
			++res;
	atomicAdd(&hits[blockIdx.x], res);
}

double GetPI(bool isEffective, unsigned long long& hits, int &dots, int &experiments, int &blocks)
{
	unsigned long long res = 0;
	unsigned long long*res_arr;
	if (!isEffective)
		hipMemcpy(&res, &hits, sizeof(unsigned long long), hipMemcpyDeviceToHost);
	else
	{
		hipHostMalloc(&res_arr, sizeof(unsigned long long) * blocks);
		hipMemcpy(res_arr, &hits, sizeof(unsigned long long) * blocks, hipMemcpyDeviceToHost);
	}
	for (int i = 0; isEffective && i < blocks; i++)
		res += res_arr[i];
	if (isEffective)
		hipHostFree(res_arr);
	return static_cast<double>(res) * 4 / dots / experiments;
}



void MonteCarlo(bool isEffective, int experiments, int &blocks, int &threads, int &dots)
{
	float elapsedTime;
	hipEvent_t start, stop;
	hipStream_t stream;
	time_t seed = time(NULL);
	hipError_t error;
	hiprandState* states;
	unsigned long long* hits;
	double res;
	if (!isEffective)
	{
		hipMalloc(&hits, sizeof(unsigned long long));
		hipMemset(hits, 0, sizeof(unsigned long long));
	}
	else
	{
		hipMalloc(&hits, sizeof(unsigned long long) * blocks);
		hipMemset(hits, 0, sizeof(unsigned long long) * blocks);
	}
	hipMalloc(&states, sizeof(hiprandState) * dots * 2);
	setStates << <blocks, threads >> > (states, dots, seed);
	if ((error = hipGetLastError()) != hipSuccess || (error = hipDeviceSynchronize()) != hipSuccess)
		std::cout << hipGetErrorString(error) << std::endl << hipGetErrorName(error) << std::endl;
	hipStreamCreate(&stream);
	hipEventCreate(&start);
	hipEventCreate(&stop);
	hipEventRecord(start, stream);
	if (!isEffective)
		ineffectiveMonteCarloCall <<<blocks, threads>>> (hits, states, experiments);
	else
		effectiveMonteCarloCall <<<blocks, threads>>> (hits, states, experiments);
	if ((error = hipGetLastError()) != hipSuccess || (error = hipDeviceSynchronize()) != hipSuccess)
		std::cout << hipGetErrorString(error) << std::endl << hipGetErrorName(error) << std::endl;
	hipEventRecord(stop, stream);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipStreamDestroy(stream);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	printf("ineffective call elapsed time = %f\tPI = %f\tDifference = %f\n",
		elapsedTime, res = GetPI(isEffective, *hits, dots, experiments, blocks), abs(res - PI_KEK));
	hipFree(states);
	hipFree(hits);
}


int main()
{
	int dots;
	int slices;
	int experiments;

	std::cout << "Enter experiments count to do not more than 100000: ";
	std::cin >> experiments;
	std::cout << std::endl;
	std::cout << "Enter slices to divided a grid with values from 0 to 127(value will be adjuted to the nearest x64 multiply number): ";
	std::cin >> slices;
	std::cout << std::endl;
	slices = slices > BLOCKS_LIMIT ? BLOCKS_LIMIT - 1 : slices;
	experiments = experiments > EXPERIMENTS_LIMIT ? EXPERIMENTS_LIMIT : experiments;
	int blocks = slices / 64 + 1;
	int threads = blocks * 64;
	dots = threads * blocks;
	std::cout << "blocks = " << blocks << "\t threads = " << threads << "\texperiments = " << experiments << std::endl;
	MonteCarlo(false, experiments, blocks, threads, dots);
	hipDeviceSynchronize();
	MonteCarlo(true, experiments, blocks, threads, dots);
	return (0);
}