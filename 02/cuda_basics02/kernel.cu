#include "hip/hip_runtime.h"
﻿#include "hip/hip_runtime.h"
#include ""
#include <iostream>
#include <math.h>
#include <stdio.h>

/// <summary>
/// 
/// </summary>
/// <param name="info_volume_multiplier"></param>
/// <param name="kind">: Host to device = 1;
/// device to host = 2;</param>
/// <returns></returns>
float copySpeed (int info_volume_multiplier, int step, int kind)
{
	float elapsedTime;
	hipEvent_t start, stop;
	hipStream_t stream;

	int *a_host, *a_dev;
	hipMalloc(&a_dev, sizeof(int) * info_volume_multiplier);
	hipHostMalloc (&a_host, sizeof (int) * info_volume_multiplier);
	for (int i = 0; i < info_volume_multiplier; i++)
		a_host[i] = INT_MAX;
	hipMemcpy (a_dev, a_host, sizeof (int) * info_volume_multiplier, hipMemcpyHostToDevice);
	hipStreamCreate(&stream);
	hipEventCreate (&start);
	hipEventCreate (&stop);
	hipEventRecord(start, stream);
	if (kind == 1)
		hipMemcpy(a_dev, a_host, sizeof(int) * info_volume_multiplier, hipMemcpyHostToDevice);
	else if (kind == 2)
		hipMemcpy (a_host, a_dev, sizeof (int) * info_volume_multiplier, hipMemcpyDeviceToHost);
	hipEventRecord(stop, stream);
	hipEventSynchronize(start);
	hipEventSynchronize(stop);
	hipEventElapsedTime(&elapsedTime, start, stop);
	hipStreamDestroy(stream);
	hipEventDestroy(start);
	hipEventDestroy(stop);
	hipHostFree(a_host);
	hipFree (a_dev);
	return (elapsedTime);
}

__global__ void calculatePI(const int segments, const float start, const float end, const float section_length, float *pi)
{
	float mid_x = start + section_length * (blockIdx.x * blockDim.x + threadIdx.x + 0.5);
	pi[blockIdx.x * 1024 + threadIdx.x] = sqrtf(1 - mid_x * mid_x);
}

double get_pi(const int& segments, const float& start, const float& end)
{
	hipError_t error;
	hipDeviceProp_t deviceProp;
	float* pi_dev;
	float* pi_host;
	int seg1024;
	int blocks;
	int threads;
	float seg_length;

	hipGetDeviceProperties (&deviceProp, 0);
	seg1024 = segments > deviceProp.maxThreadsDim[0] ? segments + 1024 - segments % 1024 : segments;
	blocks = seg1024 > deviceProp.maxThreadsDim[0] ? seg1024 / 1024 : 1;
	threads = seg1024 / blocks;
	seg_length = (end - start) / seg1024;
	hipHostMalloc (&pi_host, sizeof (float) * seg1024);
	hipMalloc(&pi_dev, sizeof(float) * seg1024);
	printf("adjusted segments = %i\nblocks = %i\n"
		"threads = %i\nsegment length = %f\n", seg1024, blocks, threads, seg_length);
	calculatePI <<<blocks, threads >>> (seg1024, start, end, seg_length, pi_dev);
	error = hipGetLastError();
	if (error != hipSuccess || (error = hipDeviceSynchronize()) != hipSuccess)
		std::cout << "There was an error in cuda device calculation" << std::endl
		<< "Error description: " << hipGetErrorString(error) << std::endl << hipGetErrorName(error) << std::endl;
	hipMemcpy(pi_host, pi_dev, sizeof(float) * seg1024, hipMemcpyDeviceToHost);
	error = hipGetLastError ();
	if (error != hipSuccess || (error = hipDeviceSynchronize ()) != hipSuccess)
		std::cout << "There was an error in cuda device calculation" << std::endl
		<< "Error description: " << hipGetErrorString (error) << std::endl << hipGetErrorName (error) << std::endl;
	hipFree(pi_dev);
	error = hipGetLastError ();
	if (error != hipSuccess || (error = hipDeviceSynchronize ()) != hipSuccess)
		std::cout << "There was an error in cuda device calculation" << std::endl
		<< "Error description: " << hipGetErrorString (error) << std::endl << hipGetErrorName (error) << std::endl;
	float sum = 0.0;
	for (int i = 0; i < seg1024; i++)
		sum += pi_host[i];
	hipHostFree(pi_host);
	return 4 * sum *seg_length;
}


__global__ void calculateDzeta(float *res, float real_part)
{
	res[blockIdx.x * 1024 + threadIdx.x] =  powf(blockIdx.x * 1024 + threadIdx.x + 1, -real_part);
}

float dzeta_f(const int& segments, const float real_part)
{
	hipError_t error;
	hipDeviceProp_t deviceProp;
	float *res_host;
	float* res_dev;
	int seg1024 = segments;
	int blocks;
	int threads;

	hipGetDeviceProperties (&deviceProp, 0);
	if (segments > deviceProp.maxThreadsDim[0])
		seg1024 += 1024 - segments % 1024;
	blocks = seg1024 > deviceProp.maxThreadsDim[0] ? seg1024 / 1024 : 1;
	threads = seg1024 / blocks;
	hipMalloc(&res_dev, sizeof(float) * seg1024);
	hipHostMalloc (&res_host, sizeof (float) * seg1024);
	calculateDzeta << <blocks, threads >> > (res_dev, real_part);
	error = hipGetLastError();
	if (error != hipSuccess || (error = hipDeviceSynchronize()) != hipSuccess)
		std::cout << "There was an error in cuda device calculation" << std::endl
		<< "Error description: " << hipGetErrorString(error) << std::endl << hipGetErrorName(error) << std::endl;
	hipMemcpy(res_host, res_dev, sizeof(float) * seg1024, hipMemcpyDeviceToHost);
	hipFree(res_dev);
	float sum = 0;
	for (int i = 0; i < seg1024; i++)
		sum += res_host[i];
	hipHostFree(res_host);
	return sum;
}

void speedTests (int count, int step)
{
	float *devToHost = new float[count];
	float* hostToDev = new float[count];
	for (int i = 0; i < count; i++)
	{
		devToHost[i] = copySpeed ((i + 1) * step, step, 2);
		hostToDev[i] = copySpeed ((i + 1) * step, step, 1);
	}

	for (int i = 0; i < count; i++)
		printf ("elem count = %d\tdevToHost=%f\thostToDev=%f\n", (i + 1) * step, devToHost[i], hostToDev[i]);
}

int main ()
{
	std::string selection;
	const float START = 0;
	const float END = 1;
	int segments;
	float argument;
	int tests;
	while (1)
	{
		std::cout << "Select exercise (1 or 2 or 3), or enter exit to quit, other input discrads: ";
		std::cin >> selection;
		if (selection == "1")
		{
			std::cout << std::endl << "Enter tests count: ";
			std::cin >> tests;
			std::cout << std::endl;
			std::cout << "Enter step: ";
			std::cin >> segments;
			std::cout << std::endl;
			speedTests (tests, segments);

		}
		else if (selection == "2")
		{
			std::cout << "Enter segments count (precision by segments of x axis): ";
			std::cin >> segments;
			std::cout << std::endl << "segments = " << segments << std::endl;
			std::cout << std::endl << "PI = " << get_pi(segments, START, END) << std::endl;
		}
		else if (selection == "3")
		{
			std::cout << "Enter elements count (precision like): ";
			std::cin >> segments;
			std::cout << std::endl << "Enter real part of argument: ";
			std::cin >> argument;
			std::cout << std::endl << "segments = " << segments << std::endl
				<< "argument = " << argument << std::endl;
			std::cout << std::endl << "res = " << dzeta_f(segments, argument) << std::endl;
		}
		else if (selection == "exit")
			break ;
		std::cout << std::endl;
	}
	return 0;
}